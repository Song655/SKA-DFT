#include <starpu.h>
#include "dft.h"

static int num_threads_per_block;

void cpu_extract_vis_codelet(void *descr[], STARPU_ATTRIBUTE_UNUSED void *arg){

    Source *sources = (Source*) STARPU_VECTOR_GET_PTR(descr[0]);
    Visibility *vis  = (Visibility*) STARPU_VECTOR_GET_PTR(descr[1]);
    Complex *vis_intensity = (Complex*) STARPU_VECTOR_GET_PTR(descr[2]);

    int num_sources = STARPU_VECTOR_GET_NX(descr[0]);
    int num_visibilities = STARPU_VECTOR_GET_NX(descr[1]);

    extract_visibilities_cpu(sources,vis,vis_intensity,num_sources,num_visibilities);
}

void cuda_extract_vis_codelet(void *descr[], STARPU_ATTRIBUTE_UNUSED void *arg){

    PRECISION3 *sources = (PRECISION3*) STARPU_VECTOR_GET_PTR(descr[0]);
    PRECISION3 *vis  = (PRECISION3*) STARPU_VECTOR_GET_PTR(descr[1]);
    PRECISION2 *vis_intensity = (PRECISION2*) STARPU_VECTOR_GET_PTR(descr[2]);

    int num_sources = STARPU_VECTOR_GET_NX(descr[0]);
    int num_visibilities = STARPU_VECTOR_GET_NX(descr[1]);

    dim3 kernel_blocks(num_threads_per_block, 1, 1);
    dim3 kernel_threads((num_visibilities + num_threads_per_block -1)/num_threads_per_block, 1, 1);

    direct_fourier_transform<<<kernel_threads, kernel_blocks,0,starpu_cuda_get_local_stream()>>>(vis, vis_intensity, num_visibilities, sources, num_sources);
    hipStreamSynchronize(starpu_cuda_get_local_stream());
} 

int starpu_launch(Config *config, Source *sources, Visibility *visibilities,Complex *vis_intensity){
    
    num_threads_per_block = config-> gpu_num_threads_per_block;
    /* start the runtime */
    int ret;
    ret = starpu_init(NULL);
    if (ret == -ENODEV)
        return 77;
    STARPU_CHECK_RETURN_VALUE(ret, "starpu_init");

    struct starpu_perfmodel extract_perf_model;
    struct starpu_codelet cl;
    starpu_data_handle_t source_handle, vis_handle, intensity_handle;

    ret = starpu_memory_pin(sources, config->num_sources*sizeof(sources[0]));
    ret += starpu_memory_pin(visibilities, config->num_visibilities*sizeof(visibilities[0]));
    ret += starpu_memory_pin(vis_intensity,config->num_visibilities*sizeof(vis_intensity[0]));
    if (ret !=0 ) return 77;

    //define perfmodel
    starpu_perfmodel_init(&extract_perf_model);
    extract_perf_model.type = STARPU_HISTORY_BASED;
    extract_perf_model.symbol = "extract_perf_model";

    //define codelet
    starpu_codelet_init(&cl);
    cl.cpu_funcs[0] = cpu_extract_vis_codelet;
    cl.cpu_funcs_name[0] = "cpu_extract_vis";
    cl.cuda_funcs[0] = cuda_extract_vis_codelet;   //cuda codelet
    cl.nbuffers = 3;
    cl.modes[0] = STARPU_R;
    cl.modes[1] = STARPU_R;
    cl.modes[2] = STARPU_W;
    //cl.model = &extract_perf_model;

    //register data
    starpu_vector_data_register(&source_handle, STARPU_MAIN_RAM, (uintptr_t)sources, config->num_sources, sizeof(sources[0]));
    starpu_vector_data_register(&vis_handle, STARPU_MAIN_RAM, (uintptr_t)visibilities, config->num_visibilities, sizeof(visibilities[0]));
    starpu_vector_data_register(&intensity_handle, STARPU_MAIN_RAM, (uintptr_t)vis_intensity, config->num_visibilities, sizeof(vis_intensity[0]));

    struct starpu_data_filter horiz;
    memset(&horiz, 0, sizeof(horiz));
    horiz.filter_func = starpu_vector_filter_block;
    horiz.nchildren = config->num_task_slices;

    starpu_data_partition(vis_handle, &horiz);
    starpu_data_partition(intensity_handle, &horiz);

    for (int i = 0; i < horiz.nchildren; i++)
    {
        struct starpu_task *task = starpu_task_create();
        task->synchronous = 0;
        task->cl = &cl;

        task->handles[0] = source_handle;
        task->handles[1] = starpu_data_get_sub_data(vis_handle, 1, i);
        task->handles[2] = starpu_data_get_sub_data(intensity_handle,1,i);

        ret = starpu_task_submit(task);
        if (ret == -ENODEV)
        {
            ret = 77;
        }
        STARPU_CHECK_RETURN_VALUE(ret, "starpu_task_submit");
        starpu_data_wont_use(starpu_data_get_sub_data(intensity_handle,1,i));
    }

    ret = starpu_task_wait_for_all(); 
    STARPU_CHECK_RETURN_VALUE(ret, "starpu_task_wait_for_all");

    starpu_data_unpartition(vis_handle, STARPU_MAIN_RAM);
    starpu_data_unpartition(intensity_handle, STARPU_MAIN_RAM);

    ret = starpu_memory_unpin(sources, config->num_sources*sizeof(sources[0]));
    ret += starpu_memory_unpin(visibilities, config->num_visibilities*sizeof(visibilities[0]));
    ret += starpu_memory_unpin(vis_intensity,config->num_visibilities*sizeof(vis_intensity[0]));
    if (ret !=0 ) return 77;

    starpu_data_unregister(source_handle);
    starpu_data_unregister(vis_handle);
    starpu_data_unregister(intensity_handle);

    starpu_shutdown();
}

