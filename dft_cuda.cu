#include "hip/hip_runtime.h"

// Copyright 2019 Adam Campbell, Seth Hall, Andrew Ensor
// Copyright 2019 High Performance Computing Research Laboratory, Auckland University of Technology (AUT)

// Redistribution and use in source and binary forms, with or without
// modification, are permitted provided that the following conditions are met:

// 1. Redistributions of source code must retain the above copyright notice,
// this list of conditions and the following disclaimer.

// 2. Redistributions in binary form must reproduce the above copyright
// notice, this list of conditions and the following disclaimer in the
// documentation and/or other materials provided with the distribution.

// 3. Neither the name of the copyright holder nor the names of its
// contributors may be used to endorse or promote products derived from this
// software without specific prior written permission.

// THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS AND CONTRIBUTORS "AS IS"
// AND ANY EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE
// IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR PURPOSE
// ARE DISCLAIMED. IN NO EVENT SHALL THE COPYRIGHT HOLDER OR CONTRIBUTORS BE
// LIABLE FOR ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL, EXEMPLARY, OR
// CONSEQUENTIAL DAMAGES (INCLUDING, BUT NOT LIMITED TO, PROCUREMENT OF
// SUBSTITUTE GOODS OR SERVICES; LOSS OF USE, DATA, OR PROFITS; OR BUSINESS
// INTERRUPTION) HOWEVER CAUSED AND ON ANY THEORY OF LIABILITY, WHETHER IN
// CONTRACT, STRICT LIABILITY, OR TORT (INCLUDING NEGLIGENCE OR OTHERWISE)
// ARISING IN ANY WAY OUT OF THE USE OF THIS SOFTWARE, EVEN IF ADVISED OF THE
// POSSIBILITY OF SUCH DAMAGE.

#include <stdio.h>
#include <stdlib.h>
#include <float.h>
#include <math.h>

#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#include <hip/hip_math_constants.h>
#include <>
#include <numeric>

#include "dft.h"

__global__ void direct_fourier_transform(const __restrict__ PRECISION3 *visibility, PRECISION2 *vis_intensity, const int vis_count, const PRECISION3 *sources, const int source_count)
{
	const int vis_indx = blockIdx.x * blockDim.x + threadIdx.x;

	if(vis_indx >= vis_count)
		return;

	PRECISION2 source_sum = MAKE_PRECISION2(0.0, 0.0);
	PRECISION term = 0.0;
	PRECISION w_correction = 0.0;
	PRECISION image_correction = 0.0;
	PRECISION theta = 0.0;
	PRECISION src_correction = 0.0;

	const PRECISION3 vis = visibility[vis_indx];
	PRECISION3 src;
	PRECISION2 theta_complex = MAKE_PRECISION2(0.0, 0.0);

	const double two_PI = HIP_PI + HIP_PI;
	// For all sources
	for(int src_indx = 0; src_indx < source_count; ++src_indx)
	{	
		src = sources[src_indx];
		
		// formula sqrt
		// term = sqrt(1.0 - (src.x * src.x) - (src.y * src.y));
		// image_correction = term;
		// w_correction = term - 1.0; 

		// approximation formula (unit test fails as less accurate)
		term = 0.5 * ((src.x * src.x) + (src.y * src.y));
		w_correction = -term;
		image_correction = 1.0 - term;

		src_correction = src.z / image_correction;

		theta = (vis.x * src.x + vis.y * src.y + vis.z * w_correction) * two_PI;
		sincos(theta, &(theta_complex.y), &(theta_complex.x));
		source_sum.x += theta_complex.x * src_correction;
		source_sum.y += -theta_complex.y * src_correction;
	}

	vis_intensity[vis_indx] = MAKE_PRECISION2(source_sum.x, source_sum.y);
}


/**
 * Check the return value of the CUDA runtime API call and exit
 * the application if the call has failed.
 */
void check_cuda_error_aux(const char *file, unsigned line, const char *statement, hipError_t err)
{
	if (err == hipSuccess)
		return;

	printf(">>> CUDA ERROR: %s returned %s at %s : %u ",statement, file, hipGetErrorString(err), line);
	exit(EXIT_FAILURE);
}

void extract_visibilities_cuda(Config *config, Source *sources, Visibility *visibilities,
	Complex *vis_intensity, int num_visibilities)
{
	//Allocating GPU memory for visibility intensity
	PRECISION3 *device_sources;
	PRECISION3 *device_visibilities;
	PRECISION2 *device_intensities;

	if(config->enable_messages)
		printf(">>> UPDATE: Allocating GPU memory...\n\n");

	//copy the sources to the GPU.
	CUDA_CHECK_RETURN(hipMalloc(&device_sources,  sizeof(PRECISION3) * config->num_sources));
	CUDA_CHECK_RETURN(hipMemcpy(device_sources, sources, 
		config->num_sources * sizeof(PRECISION3), hipMemcpyHostToDevice));
	hipDeviceSynchronize();

	//copy the visibilities to the GPU
	CUDA_CHECK_RETURN(hipMalloc(&device_visibilities,  sizeof(PRECISION3) * num_visibilities));
	CUDA_CHECK_RETURN(hipMemcpy(device_visibilities, visibilities, 
		num_visibilities * sizeof(PRECISION3), hipMemcpyHostToDevice));
	hipDeviceSynchronize();

	// Allocate memory on GPU for storing extracted visibility intensities
	CUDA_CHECK_RETURN(hipMalloc(&device_intensities,  sizeof(PRECISION2) * num_visibilities));
	hipDeviceSynchronize();

	// Define number of blocks and threads per block on GPU
        int threads_per_block = min(config->gpu_num_threads_per_block, num_visibilities);
        int num_blocks = ceil((double)num_visibilities / threads_per_block);

	dim3 kernel_threads(threads_per_block, 1, 1);
	dim3 kernel_blocks(num_blocks, 1, 1);

	if(config->enable_messages)
		printf(">>> UPDATE: Calling DFT GPU Kernel to create %d visibilities...\n\n", num_visibilities);

	//record events for timing kernel execution
	hipEvent_t start, stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);
	hipEventRecord(start);

	direct_fourier_transform<<<kernel_blocks,kernel_threads>>>(device_visibilities,
		device_intensities, num_visibilities, device_sources, config->num_sources);
	hipDeviceSynchronize();

	hipEventRecord(stop);
	hipEventSynchronize(stop);
	float milliseconds = 0;
	hipEventElapsedTime(&milliseconds, start, stop);

	if(config->enable_messages)
		printf(">>> UPDATE: DFT GPU Kernel Completed, Time taken %f mS...\n\n",milliseconds);

	CUDA_CHECK_RETURN(hipMemcpy(vis_intensity, device_intensities, 
		num_visibilities * sizeof(PRECISION2), hipMemcpyDeviceToHost));
	hipDeviceSynchronize();

	if(config->enable_messages)
		printf(">>> UPDATE: Copied Visibility Data back to Host - Completed...\n\n");

	// Clean up
	CUDA_CHECK_RETURN(hipFree(device_intensities));
	CUDA_CHECK_RETURN(hipFree(device_sources));
	CUDA_CHECK_RETURN(hipFree(device_visibilities));
	CUDA_CHECK_RETURN(hipDeviceReset());
}



